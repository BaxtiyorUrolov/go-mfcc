                                         #include <hip/hip_runtime.h>
                                           #include <hipfft/hipfft.h>

                                           // Power spectrumini hisoblash uchun CUDA kernel
                                           __global__ void powerSpectrumKernel(hipfftComplex* fftOut, float* powerSpec, int n) {
                                               int idx = blockIdx.x * blockDim.x + threadIdx.x;
                                               if (idx < n/2 + 1) {
                                                   float re = fftOut[idx].x;
                                                   float im = fftOut[idx].y;
                                                   powerSpec[idx] = re * re + im * im;
                                               }
                                           }

                                           // Mel filtrlarini qo‘llash uchun CUDA kernel
                                           __global__ void applyMelFiltersKernel(float* powerSpec, float* filterBanks, float* melEnergies, int numFilters, int frameSize) {
                                               int idx = blockIdx.x * blockDim.x + threadIdx.x;
                                               if (idx < numFilters) {
                                                   float energy = 0.0f;
                                                   for (int j = 0; j < frameSize/2 + 1; j++) {
                                                       energy += powerSpec[j] * filterBanks[idx * (frameSize/2 + 1) + j];
                                                   }
                                                   melEnergies[idx] = energy;
                                               }
                                           }

                                           // Log operatsiyasi uchun CUDA kernel
                                           __global__ void logKernel(float* input, float* output, int n) {
                                               int idx = blockIdx.x * blockDim.x + threadIdx.x;
                                               if (idx < n) {
                                                   output[idx] = logf(input[idx] + 1e-6f);
                                               }
                                           }

                                           // DCT uchun CUDA kernel
                                           __global__ void dctKernel(float* input, float* output, int n, int numCoeffs, float sqrt2OverN) {
                                               int idx = blockIdx.x * blockDim.x + threadIdx.x;
                                               if (idx < numCoeffs) {
                                                   float sum = 0.0f;
                                                   for (int m = 0; m < n; m++) {
                                                       float angle = 3.14159265359 * idx * (m + 0.5) / n;
                                                       sum += input[m] * cosf(angle);
                                                   }
                                                   output[idx] = sum * sqrt2OverN;
                                               }
                                           }

                                           // CUDA kernelni Go’dan chaqirish uchun yordamchi funksiyalar
                                           extern "C" void launchPowerSpectrumKernel(hipfftComplex* fftOut, float* powerSpec, int n, int gridSize, int blockSize, hipStream_t stream) {
                                               powerSpectrumKernel<<<gridSize, blockSize, 0, stream>>>(fftOut, powerSpec, n);
                                           }

                                           extern "C" void launchApplyMelFiltersKernel(float* powerSpec, float* filterBanks, float* melEnergies, int numFilters, int frameSize, int gridSize, int blockSize, hipStream_t stream) {
                                               applyMelFiltersKernel<<<gridSize, blockSize, 0, stream>>>(powerSpec, filterBanks, melEnergies, numFilters, frameSize);
                                           }

                                           extern "C" void launchLogKernel(float* input, float* output, int n, int gridSize, int blockSize, hipStream_t stream) {
                                               logKernel<<<gridSize, blockSize, 0, stream>>>(input, output, n);
                                           }

                                           extern "C" void launchDctKernel(float* input, float* output, int n, int numCoeffs, float sqrt2OverN, int gridSize, int blockSize, hipStream_t stream) {
                                               dctKernel<<<gridSize, blockSize, 0, stream>>>(input, output, n, numCoeffs, sqrt2OverN);
                                           }